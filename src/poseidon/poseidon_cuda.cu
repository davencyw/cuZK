#include "hip/hip_runtime.h"
#include "poseidon_cuda.cuh"
#include "field_arithmetic_cuda.cuh"  // Still needed for class methods like initialize()
#include "cuda_field_element.cuh"
#include "../common/error_handling.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <chrono>
#include <iostream>

using namespace Poseidon::CudaFieldOps;  // For CudaFieldArithmetic class methods and CudaFieldElement arithmetic functions

namespace Poseidon {
namespace PoseidonCUDA {

// Use standardized CUDA error handling - these are macros, imported via include
using cuZK::ErrorHandling::cuda_sync_check;

// Static member initialization
bool CudaPoseidonHash::initialized_ = false;
size_t CudaPoseidonHash::optimal_batch_size_ = 1024;
size_t CudaPoseidonHash::max_batch_size_ = 65536;
FieldElement* CudaPoseidonHash::d_round_constants_ = nullptr;
FieldElement* CudaPoseidonHash::d_mds_matrix_ = nullptr;

// Device pointers for constants (initialized at runtime)
__device__ FieldElement* d_poseidon_round_constants_ptr;
__device__ FieldElement* d_poseidon_mds_matrix_ptr;

// ================================
// Device Functions for Poseidon Operations
// ================================

__device__ void cuda_add_round_constants(CudaFieldElement state[PoseidonParams::STATE_SIZE], size_t round) {
    for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
        size_t const_idx = round * PoseidonParams::STATE_SIZE + i;
        CudaFieldElement round_const = CudaFieldElement(
            d_poseidon_round_constants_ptr[const_idx].limbs[0],
            d_poseidon_round_constants_ptr[const_idx].limbs[1],
            d_poseidon_round_constants_ptr[const_idx].limbs[2],
            d_poseidon_round_constants_ptr[const_idx].limbs[3]
        );
        cuda_add(state[i], round_const, state[i]);
    }
}

__device__ void cuda_apply_sbox(CudaFieldElement state[PoseidonParams::STATE_SIZE]) {
    for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
        cuda_power5(state[i], state[i]);
    }
}

__device__ void cuda_apply_partial_sbox(CudaFieldElement state[PoseidonParams::STATE_SIZE]) {
    // Only apply S-box to the first element in partial rounds
    cuda_power5(state[0], state[0]);
}

__device__ void cuda_apply_mds_matrix(CudaFieldElement state[PoseidonParams::STATE_SIZE]) {
    // Store original state and initialize new state
    CudaFieldElement original_state[PoseidonParams::STATE_SIZE];
    CudaFieldElement new_state[PoseidonParams::STATE_SIZE];
    
    // Copy original state and initialize new state to zero
    for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
        original_state[i] = state[i];
        new_state[i].set_zero();
    }
    
    // Compute MDS matrix multiplication: new_state[i] = sum(MDS[i][j] * original_state[j])
    for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
        for (size_t j = 0; j < PoseidonParams::STATE_SIZE; ++j) {
            size_t matrix_idx = i * PoseidonParams::STATE_SIZE + j;
            CudaFieldElement mds_element = CudaFieldElement(
                d_poseidon_mds_matrix_ptr[matrix_idx].limbs[0],
                d_poseidon_mds_matrix_ptr[matrix_idx].limbs[1],
                d_poseidon_mds_matrix_ptr[matrix_idx].limbs[2],
                d_poseidon_mds_matrix_ptr[matrix_idx].limbs[3]
            );
            
            // Multiply: product = MDS[i][j] * original_state[j]
            CudaFieldElement product;
            cuda_multiply(mds_element, original_state[j], product);
            
            // Add to accumulator: new_state[i] += product
            cuda_add(new_state[i], product, new_state[i]);
        }
    }
    
    // Copy final result back to state
    for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
        state[i] = new_state[i];
    }
}

__device__ void cuda_permutation(CudaFieldElement state[PoseidonParams::STATE_SIZE]) {
    size_t round = 0;
    
    // First half of full rounds
    for (size_t r = 0; r < PoseidonParams::ROUNDS_FULL / 2; ++r) {
        cuda_add_round_constants(state, round++);
        cuda_apply_sbox(state);
        cuda_apply_mds_matrix(state);
    }
    
    // Partial rounds
    for (size_t r = 0; r < PoseidonParams::ROUNDS_PARTIAL; ++r) {
        cuda_add_round_constants(state, round++);
        cuda_apply_partial_sbox(state);
        cuda_apply_mds_matrix(state);
    }
    
    // Second half of full rounds
    for (size_t r = 0; r < PoseidonParams::ROUNDS_FULL / 2; ++r) {
        cuda_add_round_constants(state, round++);
        cuda_apply_sbox(state);
        cuda_apply_mds_matrix(state);
    }
}

// ================================
// Device Hash Functions for Merkle Trees
// ================================

__device__ CudaFieldElement device_hash_n(const CudaFieldElement* children, size_t arity) {
    // Initialize with domain separator for sponge construction
    CudaFieldElement state[PoseidonParams::STATE_SIZE];
    state[0] = CudaFieldElement(3); // Same domain separator as CPU hash_multiple
    state[1] = CudaFieldElement(0); 
    state[2] = CudaFieldElement(0);
    
    // Absorb phase - process children in chunks
    size_t input_idx = 0;
    while (input_idx < arity) {
        // Add up to RATE children to the state (rate = 2, capacity = 1)
        for (size_t i = 0; i < PoseidonParams::RATE && input_idx < arity; ++i) {
            // Add to state[i + CAPACITY] = state[i + 1]
            cuda_add(state[i + PoseidonParams::CAPACITY], children[input_idx], 
                    state[i + PoseidonParams::CAPACITY]);
            input_idx++;
        }
        
        // Apply permutation
        cuda_permutation(state);
    }
    
    // Squeeze phase - return first rate element (state[CAPACITY] = state[1])
    return state[PoseidonParams::CAPACITY];
}

// ================================
// Kernel Functions
// ================================

__global__ void batch_hash_single_kernel(const CudaFieldElement* inputs, CudaFieldElement* outputs, size_t count) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < count) {
        // Initialize Poseidon state: [1, input, 0]
        CudaFieldElement state[PoseidonParams::STATE_SIZE];
        state[0] = CudaFieldElement(1); // Domain separator for single hash
        state[1] = inputs[idx];         // Input
        state[2] = CudaFieldElement(0); // Zero padding
        
        // Apply full Poseidon permutation
        cuda_permutation(state);
        
        // Store result (first rate element)
        outputs[idx] = state[1];
    }
}

__global__ void batch_hash_pairs_kernel(const CudaFieldElement* left_inputs, const CudaFieldElement* right_inputs, CudaFieldElement* outputs, size_t count) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < count) {
        // Initialize Poseidon state: [2, left, right]
        CudaFieldElement state[PoseidonParams::STATE_SIZE];
        state[0] = CudaFieldElement(2);     // Domain separator for pair hash
        state[1] = left_inputs[idx];        // Left input
        state[2] = right_inputs[idx];       // Right input
        
        // Apply full Poseidon permutation
        cuda_permutation(state);
        
        // Store result (first rate element)
        outputs[idx] = state[1];
    }
}

__global__ void batch_permutation_kernel(CudaFieldElement* states, size_t count) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < count) {
        // Apply permutation to the state at index idx
        // Each state consists of PoseidonParams::STATE_SIZE (3) CudaFieldElements
        size_t state_offset = idx * PoseidonParams::STATE_SIZE;
        
        // Extract the state for this thread - now using natural assignment
        CudaFieldElement local_state[PoseidonParams::STATE_SIZE];
        for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
            local_state[i] = states[state_offset + i];
        }
        
        // Apply the Poseidon permutation
        cuda_permutation(local_state);
        
        // Write the result back - natural assignment
        for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
            states[state_offset + i] = local_state[i];
        }
    }
}

// ================================
// Host Interface Implementation
// ================================

bool CudaPoseidonHash::initialize() {
    if (initialized_) {
        return true;
    }
    
    // Initialize CUDA field arithmetic first
    if (!CudaFieldArithmetic::initialize()) {
        std::cerr << "Failed to initialize CUDA field arithmetic" << std::endl;
        return false;
    }
    
    // Initialize Poseidon constants on host
    PoseidonConstants::init();
    
    // Copy constants to device
    if (!copy_constants_to_device()) {
        std::cerr << "Failed to copy Poseidon constants to device" << std::endl;
        return false;
    }
    
    // Determine optimal batch sizes based on device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    optimal_batch_size_ = std::min(static_cast<size_t>(prop.maxThreadsPerBlock), static_cast<size_t>(1024));
    max_batch_size_ = std::min(static_cast<size_t>(prop.maxGridSize[0] * optimal_batch_size_), static_cast<size_t>(1 << 20)); // 1M max
    
    initialized_ = true;
    return true;
}

void CudaPoseidonHash::cleanup() {
    if (d_round_constants_) {
        hipFree(d_round_constants_);
        d_round_constants_ = nullptr;
    }
    
    if (d_mds_matrix_) {
        hipFree(d_mds_matrix_);
        d_mds_matrix_ = nullptr;
    }
    
    CudaFieldArithmetic::cleanup();
    initialized_ = false;
}

bool CudaPoseidonHash::copy_constants_to_device() {
    // Allocate device memory for round constants
    size_t round_constants_size = PoseidonParams::TOTAL_ROUNDS * PoseidonParams::STATE_SIZE * sizeof(FieldElement);
    CUDA_CHECK_RETURN(hipMalloc(&d_round_constants_, round_constants_size));
    
    // Copy round constants to device
    CUDA_CHECK_RETURN(hipMemcpy(d_round_constants_, PoseidonConstants::ROUND_CONSTANTS.data(), round_constants_size, hipMemcpyHostToDevice));
    
    // Allocate device memory for MDS matrix
    size_t mds_matrix_size = PoseidonParams::STATE_SIZE * PoseidonParams::STATE_SIZE * sizeof(FieldElement);
    CUDA_CHECK_RETURN(hipMalloc(&d_mds_matrix_, mds_matrix_size));
    
    // Copy MDS matrix to device
    CUDA_CHECK_RETURN(hipMemcpy(d_mds_matrix_, PoseidonConstants::MDS_MATRIX.data(), mds_matrix_size, hipMemcpyHostToDevice));
    
    // Copy device pointers to device symbol memory (for inline functions)
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_poseidon_round_constants_ptr), &d_round_constants_, sizeof(FieldElement*)));
    
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_poseidon_mds_matrix_ptr), &d_mds_matrix_, sizeof(FieldElement*)));
    
    return true;
}

bool CudaPoseidonHash::batch_hash_single(const std::vector<FieldElement>& inputs, std::vector<FieldElement>& outputs) {
    if (!initialized_) {
        std::cerr << "CudaPoseidonHash not initialized" << std::endl;
        return false;
    }
    
    size_t count = inputs.size();
    outputs.resize(count);
    
    if (count == 0) return true;
    
    // Convert inputs to CudaFieldElement
    std::vector<CudaFieldElement> cuda_inputs;
    cuda_inputs.reserve(count);
    for (const auto& fe : inputs) {
        cuda_inputs.emplace_back(fe);
    }
    
    // Allocate device memory for CudaFieldElement
    CudaFieldElement* d_inputs;
    CudaFieldElement* d_outputs;
    
    CUDA_CHECK_RETURN(hipMalloc(&d_inputs, count * sizeof(CudaFieldElement)));
    
    CUDA_CHECK_RETURN(hipMalloc(&d_outputs, count * sizeof(CudaFieldElement)));
    
    // Copy input data to device
    CUDA_CHECK_RETURN(hipMemcpy(d_inputs, cuda_inputs.data(), count * sizeof(CudaFieldElement), hipMemcpyHostToDevice));
    
    // Configure kernel launch parameters
    // Use smaller block size for Poseidon kernels due to high register usage
    size_t block_size = std::min(size_t(128), std::min(optimal_batch_size_, count));
    size_t grid_size = (count + block_size - 1) / block_size;
    
    // Launch kernel
    batch_hash_single_kernel<<<grid_size, block_size>>>(d_inputs, d_outputs, count);
    
    CUDA_KERNEL_CHECK();

    // Wait for kernel completion
    cuda_sync_check();
    
    // Copy results back to host and convert to FieldElement
    std::vector<CudaFieldElement> cuda_outputs(count);
    CUDA_CHECK_RETURN(hipMemcpy(cuda_outputs.data(), d_outputs, count * sizeof(CudaFieldElement), hipMemcpyDeviceToHost));
    
    // Convert back to FieldElement
    for (size_t i = 0; i < count; ++i) {
        outputs[i] = FieldElement(cuda_outputs[i].limbs[0], cuda_outputs[i].limbs[1], 
                                cuda_outputs[i].limbs[2], cuda_outputs[i].limbs[3]);
    }
    
    // Clean up device memory
    hipFree(d_inputs);
    hipFree(d_outputs);
    
    return true;
}

bool CudaPoseidonHash::batch_hash_pairs(const std::vector<FieldElement>& left_inputs,
                                       const std::vector<FieldElement>& right_inputs,
                                       std::vector<FieldElement>& outputs) {
    if (!initialized_) {
        std::cerr << "CudaPoseidonHash not initialized" << std::endl;
        return false;
    }
    
    if (left_inputs.size() != right_inputs.size()) {
        std::cerr << "Left and right input vectors must have the same size" << std::endl;
        return false;
    }
    
    size_t count = left_inputs.size();
    outputs.resize(count);
    
    if (count == 0) return true;
    
    // Convert inputs to CudaFieldElement
    std::vector<CudaFieldElement> cuda_left_inputs;
    std::vector<CudaFieldElement> cuda_right_inputs;
    cuda_left_inputs.reserve(count);
    cuda_right_inputs.reserve(count);
    
    for (const auto& fe : left_inputs) {
        cuda_left_inputs.emplace_back(fe);
    }
    for (const auto& fe : right_inputs) {
        cuda_right_inputs.emplace_back(fe);
    }
    
    // Allocate device memory for CudaFieldElement
    CudaFieldElement* d_left_inputs;
    CudaFieldElement* d_right_inputs;
    CudaFieldElement* d_outputs;
    
    CUDA_CHECK_RETURN(hipMalloc(&d_left_inputs, count * sizeof(CudaFieldElement)));
    CUDA_CHECK_RETURN(hipMalloc(&d_right_inputs, count * sizeof(CudaFieldElement)));
    CUDA_CHECK_RETURN(hipMalloc(&d_outputs, count * sizeof(CudaFieldElement)));
    
    // Copy input data to device
    CUDA_CHECK_RETURN(hipMemcpy(d_left_inputs, cuda_left_inputs.data(), count * sizeof(CudaFieldElement), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_right_inputs, cuda_right_inputs.data(), count * sizeof(CudaFieldElement), hipMemcpyHostToDevice));
    
    // Configure kernel launch parameters
    // Use smaller block size for Poseidon kernels due to high register usage
    size_t block_size = std::min(size_t(128), std::min(optimal_batch_size_, count));
    size_t grid_size = (count + block_size - 1) / block_size;
    
    // Launch kernel
    batch_hash_pairs_kernel<<<grid_size, block_size>>>(d_left_inputs, d_right_inputs, d_outputs, count);
    
    CUDA_KERNEL_CHECK();
    
    // Wait for kernel completion
    cuda_sync_check();
    
    // Copy results back to host and convert to FieldElement
    std::vector<CudaFieldElement> cuda_outputs(count);
    CUDA_CHECK_RETURN(hipMemcpy(cuda_outputs.data(), d_outputs, count * sizeof(CudaFieldElement), hipMemcpyDeviceToHost));
    
    // Convert back to FieldElement
    for (size_t i = 0; i < count; ++i) {
        outputs[i] = FieldElement(cuda_outputs[i].limbs[0], cuda_outputs[i].limbs[1], 
                                cuda_outputs[i].limbs[2], cuda_outputs[i].limbs[3]);
    }
    
    // Clean up device memory
    hipFree(d_left_inputs);
    hipFree(d_right_inputs);
    hipFree(d_outputs);
    
    return true;
}

bool CudaPoseidonHash::gpu_hash_single(const FieldElement& input, FieldElement& output) {
    std::vector<FieldElement> inputs = {input};
    std::vector<FieldElement> outputs;
    
    bool success = batch_hash_single(inputs, outputs);
    if (success && !outputs.empty()) {
        output = outputs[0];
    }
    
    return success;
}

bool CudaPoseidonHash::gpu_hash_pair(const FieldElement& left, const FieldElement& right, FieldElement& output) {
    std::vector<FieldElement> left_inputs = {left};
    std::vector<FieldElement> right_inputs = {right};
    std::vector<FieldElement> outputs;
    
    bool success = batch_hash_pairs(left_inputs, right_inputs, outputs);
    if (success && !outputs.empty()) {
        output = outputs[0];
    }
    
    return success;
}

bool CudaPoseidonHash::batch_permutation(std::vector<std::array<CudaFieldElement, PoseidonParams::STATE_SIZE>>& states) {
    if (!initialized_) {
        std::cerr << "CudaPoseidonHash not initialized" << std::endl;
        return false;
    }
    
    size_t count = states.size();
    if (count == 0) return true;
    
    // Flatten the states array for GPU processing
    std::vector<CudaFieldElement> flattened_states;
    flattened_states.reserve(count * PoseidonParams::STATE_SIZE);
    
    for (const auto& state : states) {
        for (size_t i = 0; i < PoseidonParams::STATE_SIZE; ++i) {
            flattened_states.push_back(state[i]);
        }
    }
    
    // Allocate device memory
    CudaFieldElement* d_states;
    CUDA_CHECK_RETURN(hipMalloc(&d_states, count * PoseidonParams::STATE_SIZE * sizeof(CudaFieldElement)));
    
    // Copy data to device
    CUDA_CHECK_RETURN(hipMemcpy(d_states, flattened_states.data(), 
                       count * PoseidonParams::STATE_SIZE * sizeof(CudaFieldElement), 
                       hipMemcpyHostToDevice));
    
    // Configure kernel launch parameters
    // Use smaller block size for Poseidon kernels due to high register usage
    size_t block_size = std::min(size_t(128), std::min(optimal_batch_size_, count));
    size_t grid_size = (count + block_size - 1) / block_size;
    
    // Launch kernel
    batch_permutation_kernel<<<grid_size, block_size>>>(d_states, count);
    
    CUDA_KERNEL_CHECK();
    
    // Wait for kernel completion
    cuda_sync_check();
    
    // Copy results back to host
    std::vector<CudaFieldElement> result_flattened(count * PoseidonParams::STATE_SIZE);
    CUDA_CHECK_RETURN(hipMemcpy(result_flattened.data(), d_states, 
                       count * PoseidonParams::STATE_SIZE * sizeof(CudaFieldElement), 
                       hipMemcpyDeviceToHost));
    
    // Unflatten the results back into the states array
    for (size_t i = 0; i < count; ++i) {
        for (size_t j = 0; j < PoseidonParams::STATE_SIZE; ++j) {
            states[i][j] = result_flattened[i * PoseidonParams::STATE_SIZE + j];
        }
    }
    
    // Clean up device memory
    hipFree(d_states);
    
    return true;
}

// ================================
// Utility Functions
// ================================

size_t CudaPoseidonHash::get_optimal_batch_size() {
    return optimal_batch_size_;
}

size_t CudaPoseidonHash::get_max_batch_size() {
    return max_batch_size_;
}

void CudaPoseidonHash::print_performance_info() {
    if (!initialized_) {
        std::cout << "CudaPoseidonHash not initialized" << std::endl;
        return;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    std::cout << "CUDA Poseidon Performance Info:" << std::endl;
    std::cout << "  Device: " << prop.name << std::endl;
    std::cout << "  Optimal batch size: " << optimal_batch_size_ << std::endl;
    std::cout << "  Maximum batch size: " << max_batch_size_ << std::endl;
    std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
    std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "  Global memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
}

// Memory management functions for FieldElement - kept for existing kernels that still use FieldElement
FieldElement* CudaPoseidonHash::allocate_device_memory(size_t count) {
    FieldElement* ptr;
    CUDA_MALLOC_CHECK(ptr, count * sizeof(FieldElement));
    return ptr;
}

void CudaPoseidonHash::free_device_memory(FieldElement* ptr) {
    if (ptr) {
        hipFree(ptr);
    }
}

bool CudaPoseidonHash::copy_to_device(const std::vector<FieldElement>& host_data, FieldElement* device_ptr) {
    CUDA_CHECK_RETURN(hipMemcpy(device_ptr, host_data.data(), 
                                 host_data.size() * sizeof(FieldElement), 
                                 hipMemcpyHostToDevice));
    return true;
}

bool CudaPoseidonHash::copy_from_device(FieldElement* device_ptr, std::vector<FieldElement>& host_data, size_t count) {
    CUDA_CHECK_RETURN(hipMemcpy(host_data.data(), device_ptr, 
                                 count * sizeof(FieldElement), 
                                 hipMemcpyDeviceToHost));
    return true;
}



// ================================
// Benchmark Functions
// ================================

CudaPoseidonStats benchmark_cuda_poseidon_single(size_t num_hashes, size_t batch_size) {
    CudaPoseidonStats stats = {};
    
    if (!CudaPoseidonHash::initialize()) {
        std::cerr << "Failed to initialize CUDA Poseidon" << std::endl;
        return stats;
    }
    
    // Generate test data
    std::vector<FieldElement> inputs;
    inputs.reserve(num_hashes);
    for (size_t i = 0; i < num_hashes; ++i) {
        inputs.push_back(FieldElement::random());
    }
    
    std::vector<FieldElement> outputs;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // Process in batches
    for (size_t i = 0; i < num_hashes; i += batch_size) {
        size_t current_batch_size = std::min(batch_size, num_hashes - i);
        std::vector<FieldElement> batch_inputs(inputs.begin() + i, inputs.begin() + i + current_batch_size);
        std::vector<FieldElement> batch_outputs;
        
        if (!CudaPoseidonHash::batch_hash_single(batch_inputs, batch_outputs)) {
            std::cerr << "Failed to process batch at index " << i << std::endl;
            return stats;
        }
        
        outputs.insert(outputs.end(), batch_outputs.begin(), batch_outputs.end());
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    
    stats.total_time_ms = duration.count() / 1000000.0;
    stats.avg_time_per_hash_ns = static_cast<double>(duration.count()) / num_hashes;
    stats.hashes_per_second = static_cast<size_t>(1000000000.0 / stats.avg_time_per_hash_ns);
    stats.total_hashes = num_hashes;
    
    // Estimate GPU memory usage
    stats.gpu_memory_used_mb = (batch_size * 2 * sizeof(FieldElement)) / (1024 * 1024); // inputs + outputs
    
    return stats;
}

CudaPoseidonStats benchmark_cuda_poseidon_pairs(size_t num_pairs, size_t batch_size) {
    CudaPoseidonStats stats = {};
    
    if (!CudaPoseidonHash::initialize()) {
        std::cerr << "Failed to initialize CUDA Poseidon" << std::endl;
        return stats;
    }
    
    // Generate test data
    std::vector<FieldElement> left_inputs, right_inputs;
    left_inputs.reserve(num_pairs);
    right_inputs.reserve(num_pairs);
    
    for (size_t i = 0; i < num_pairs; ++i) {
        left_inputs.push_back(FieldElement::random());
        right_inputs.push_back(FieldElement::random());
    }
    
    std::vector<FieldElement> outputs;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // Process in batches
    for (size_t i = 0; i < num_pairs; i += batch_size) {
        size_t current_batch_size = std::min(batch_size, num_pairs - i);
        std::vector<FieldElement> batch_left(left_inputs.begin() + i, left_inputs.begin() + i + current_batch_size);
        std::vector<FieldElement> batch_right(right_inputs.begin() + i, right_inputs.begin() + i + current_batch_size);
        std::vector<FieldElement> batch_outputs;
        
        if (!CudaPoseidonHash::batch_hash_pairs(batch_left, batch_right, batch_outputs)) {
            std::cerr << "Failed to process batch at index " << i << std::endl;
            return stats;
        }
        
        outputs.insert(outputs.end(), batch_outputs.begin(), batch_outputs.end());
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    
    stats.total_time_ms = duration.count() / 1000000.0;
    stats.avg_time_per_hash_ns = static_cast<double>(duration.count()) / num_pairs;
    stats.hashes_per_second = static_cast<size_t>(1000000000.0 / stats.avg_time_per_hash_ns);
    stats.total_hashes = num_pairs;
    
    // Estimate GPU memory usage
    stats.gpu_memory_used_mb = (batch_size * 3 * sizeof(FieldElement)) / (1024 * 1024); // left + right + outputs
    
    return stats;
}

CudaPoseidonStats benchmark_cuda_vs_cpu_poseidon(size_t num_hashes, size_t batch_size) {
    CudaPoseidonStats stats = benchmark_cuda_poseidon_single(num_hashes, batch_size);
    
    // Benchmark CPU version for comparison
    auto cpu_stats = Poseidon::benchmark_poseidon(num_hashes);
    
    if (cpu_stats.avg_time_per_hash_ns > 0) {
        stats.speedup_vs_cpu = cpu_stats.avg_time_per_hash_ns / stats.avg_time_per_hash_ns;
    }
    
    return stats;
}

} // namespace PoseidonCUDA
} // namespace Poseidon 